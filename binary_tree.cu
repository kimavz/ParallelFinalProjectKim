
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void build_binary_tree(int *x, int *child, int *root, int n)
{
	int bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	int offset = 0;
	bool newBody = true;
	int rootValue = *root;

	// build binary tree
	int childPath;
	int temp;
	offset = 0;
	while((bodyIndex + offset) < n){

		if(newBody){
			newBody = false;

			temp = 0;
			childPath = 0;
			if(x[bodyIndex + offset] > rootValue){
				childPath = 1;
			}
		}
		int childIndex = child[temp*2 + childPath];

		// traverse tree until we hit leaf node
		while(childIndex >= 0){
			temp = childIndex;
			childPath = 0;
			if(x[bodyIndex + offset] > temp){
				childPath = 1;
			}

			childIndex = child[2*temp + childPath];
		}


		if(childIndex != -2){
			int locked = temp*2 + childPath;
			if(atomicCAS(&child[locked], childIndex, -2) == childIndex){
				if(childIndex == -1){
					child[locked] = x[bodyIndex + offset];
				}

				offset += stride;
				newBody = true;
			}
		}

		__syncthreads(); // not strictly needed
	}
}

int main(){
	int n = 32;

	int *h_x;	//host array
	int *d_x;	//device array
	int *h_root;
	int *d_root;
	int *h_child;
	int *d_child;

	// allocate memory
	h_x = (int*)malloc(n*sizeof(int));
	h_root = (int*)malloc(sizeof(int));
	h_child = (int*)malloc(2*(n+1)*sizeof(int));
	hipMalloc((void**)&d_root, sizeof(int));
	hipMalloc((void**)&d_x, n*sizeof(int));
	hipMalloc((void**)&d_child, 2*(n+1)*sizeof(int));
	hipMemset(d_child, -1, 2*(n+1)*sizeof(int));


	// fill h_temp and h_x arrays
	for(int i=0;i<n;i++){
		h_x[i] = i+1;
	}

	// shuffling the array
	for(int i=0;i<n;i++){
		int j = random() % (n-i);
		int temp = h_x[i];
		h_x[i] = h_x[i+j];
		h_x[i+j] = temp;
	}
	*h_root = h_x[0];

	for(int i=0;i<n;i++){
		printf("%d ", h_x[i]);
	}
	printf("\n");


	// copy data to device
	hipMemcpy(d_root, h_root, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_x, h_x, n*sizeof(int), hipMemcpyHostToDevice);


	// kernel call
	build_binary_tree<<< 16, 16>>>(d_x, d_child, d_root, n);


	// copy from device back to host
	hipMemcpy(h_child, d_child, 2*(n+1)*sizeof(int), hipMemcpyDeviceToHost);


	// print tree
	for(int i=0;i<2*(n+1);i++){
		printf("%d ", h_child[i]);
	}
	printf("\n");

	// free memory
	free(h_x);
	free(h_root);
	free(h_child);
	hipFree(d_x);
	hipFree(d_root);
	hipFree(d_child);
}
